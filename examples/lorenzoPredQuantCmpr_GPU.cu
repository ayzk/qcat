#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rw.h"
#include "sz_utility.h"

struct PrivateTimingGPU {
    hipEvent_t start;
    hipEvent_t stop;
};

class TimingGPU
{
    private:
        PrivateTimingGPU *privateTimingGPU;

    public:

        TimingGPU();

        ~TimingGPU();

        void StartCounter();
        void StartCounterFlags();

        float GetCounter();

}; // TimingGPU class

// default constructor
TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU;  }

// default destructor
TimingGPU::~TimingGPU() { }

void TimingGPU::StartCounter()
{
    hipEventCreate(&((*privateTimingGPU).start));
    hipEventCreate(&((*privateTimingGPU).stop));
    hipEventRecord((*privateTimingGPU).start,0);
}

void TimingGPU::StartCounterFlags()
{
    int eventflags = hipEventBlockingSync;

    hipEventCreateWithFlags(&((*privateTimingGPU).start),eventflags);
    hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventflags);
    hipEventRecord((*privateTimingGPU).start,0);
}

// Gets the counter in ms
float TimingGPU::GetCounter()
{
    float time;
    hipEventRecord((*privateTimingGPU).stop, 0);
    hipEventSynchronize((*privateTimingGPU).stop);
    hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
    return time;
}


__device__ inline int kernel_quantization_float(float data, float recipPrecision)
{
    float dataRecip = data*recipPrecision;
    int s = dataRecip>=-0.5f?0:1;
    return (int)(dataRecip+0.5f) - s;
}

__device__ inline int kernel_quantization_double(double data, double recipPrecision)
{
    double dataRecip = data*recipPrecision;
    int s = dataRecip>=-0.5?0:1;
    return (int)(dataRecip+0.5) - s;
}


__global__ void kernel_lorenzoPredictorQuant_Cmpr_original_float(float* oriData, int* quantArray, float recipPrecision, int bunch)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * bunch;
    int currQuant, pre1Quant;
    int tempIdx;

    // Quantization and 1-layer Lorenzo, original mode.
    pre1Quant = index==0 ? 0 : kernel_quantization_float(oriData[index-1], recipPrecision);
    for(int i=0; i<bunch; i++)
    {
        tempIdx = index + i;
        currQuant = kernel_quantization_float(oriData[tempIdx], recipPrecision);
        quantArray[tempIdx] = currQuant - pre1Quant;
        pre1Quant = currQuant;
    }
}


__global__ void kernel_lorenzoPredictorQuant_Cmpr_original_double(double* oriData, int* quantArray, double recipPrecision, int bunch)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * bunch;
    int currQuant, pre1Quant;
    int tempIdx;

    // Quantization and 1-layer Lorenzo, original mode.
    pre1Quant = index==0 ? 0 : kernel_quantization_double(oriData[index-1], recipPrecision);
    for(int i=0; i<bunch; i++)
    {
        tempIdx = index + i;
        currQuant = kernel_quantization_double(oriData[tempIdx], recipPrecision);
        quantArray[tempIdx] = currQuant - pre1Quant;
        pre1Quant = currQuant;
    }
}


__global__ void kernel_lorenzoPredictorQuant_Cmpr_normalized_float(float* oriData, int* quantArray, float recipPrecision, int bunch)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * bunch;
    int currQuant, pre1Quant, x;
    int tempIdx;

    // Quantization and 1-layer Lorenzo, normalized mode              .
    pre1Quant = index==0 ? 0 : kernel_quantization_float(oriData[index-1], recipPrecision);
    for(int i=0; i<bunch; i++)
    {
        tempIdx = index + i;
        currQuant = kernel_quantization_float(oriData[tempIdx], recipPrecision);
        x = currQuant - pre1Quant;
        quantArray[tempIdx] = (x<<1)^(x>>31);
        pre1Quant = currQuant;
    }
}


__global__ void kernel_lorenzoPredictorQuant_Cmpr_normalized_double(double* oriData, int* quantArray, double recipPrecision, int bunch)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * bunch;
    int currQuant, pre1Quant, x;
    int tempIdx;

    // Quantization and 1-layer Lorenzo, normalized mode              .
    pre1Quant = index==0 ? 0 : kernel_quantization_double(oriData[index-1], recipPrecision);
    for(int i=0; i<bunch; i++)
    {
        tempIdx = index + i;
        currQuant = kernel_quantization_double(oriData[tempIdx], recipPrecision);
        x = currQuant - pre1Quant;
        quantArray[tempIdx] = (x<<1)^(x>>31);
        pre1Quant = currQuant;
    }
}

int lorenzoPredictorQuant_Cmpr_NoOutlier_GPU_float(float* data, int codeFormat, double errorBound, size_t n3, size_t n2, size_t n1, int* out)
{
    TimingGPU timer_GPU;
    size_t nbEle = computeDataLength(0, 0, n3, n2, n1);
    float recipPrecision = 0.5f/errorBound;
    int bsize = 256, bunch = 2;
    int gsize = nbEle / (bsize * bunch) + (nbEle % (bsize * bunch) ==0 ? 0 : 1);
    int pad_nbEle = gsize * bsize * bunch;
    float* oriData = (float*)malloc(sizeof(float)*pad_nbEle);

    memcpy(oriData, data, sizeof(float)*pad_nbEle);

    float* d_oriData;
    int* d_quantArray;

    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMalloc((void**)&d_quantArray, sizeof(int)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);

    dim3 blockSize(bsize);
    dim3 gridSize(gsize);

    timer_GPU.StartCounter(); // set timer
    if(codeFormat == QUANT_CODE_ORIGINAL)
        kernel_lorenzoPredictorQuant_Cmpr_original_float<<<gridSize, blockSize>>>(d_oriData, d_quantArray, recipPrecision, bunch);
    else if(codeFormat == QUANT_CODE_NORMALIZE)
        kernel_lorenzoPredictorQuant_Cmpr_normalized_float<<<gridSize, blockSize>>>(d_oriData, d_quantArray, recipPrecision, bunch);
    else
    {   
        // Temporarily leaving blank for other implementation.
    }
    printf("lorenzoPredQuantCmpr-float kernel speed: %f GB/s\n", (nbEle*sizeof(float)/1024.0/1024.0)/timer_GPU.GetCounter()); // print speed

    hipMemcpy(out, d_quantArray, sizeof(int)*nbEle, hipMemcpyDeviceToHost);

    free(oriData);
    hipFree(d_oriData);
    hipFree(d_quantArray);
    return 0;
}

int lorenzoPredictorQuant_Cmpr_NoOutlier_GPU_double(double* data, int codeFormat, double errorBound, size_t n3, size_t n2, size_t n1, int* out)
{
    TimingGPU timer_GPU;
    size_t nbEle = computeDataLength(0, 0, n3, n2, n1);
    double recipPrecision = 0.5/errorBound;
    int bsize = 256, bunch = 2;
    int gsize = nbEle / (bsize * bunch) + (nbEle % (bsize * bunch) ==0 ? 0 : 1);
    int pad_nbEle = gsize * bsize * bunch;
    double* oriData = (double*)malloc(sizeof(double)*pad_nbEle);

    memcpy(oriData, data, sizeof(double)*pad_nbEle);

    double* d_oriData;
    int* d_quantArray;

    hipMalloc((void**)&d_oriData, sizeof(double)*pad_nbEle);
    hipMalloc((void**)&d_quantArray, sizeof(int)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(double)*pad_nbEle, hipMemcpyHostToDevice);

    dim3 blockSize(bsize);
    dim3 gridSize(gsize);

    timer_GPU.StartCounter(); // set timer
    if(codeFormat == QUANT_CODE_ORIGINAL)
        kernel_lorenzoPredictorQuant_Cmpr_original_double<<<gridSize, blockSize>>>(d_oriData, d_quantArray, recipPrecision, bunch);
    else if(codeFormat == QUANT_CODE_NORMALIZE)
        kernel_lorenzoPredictorQuant_Cmpr_normalized_double<<<gridSize, blockSize>>>(d_oriData, d_quantArray, recipPrecision, bunch);
    else
    {   
        // Temporarily leaving blank for other implementation.
    }
    printf("lorenzoPredQuantCmpr-double kernel speed: %f GB/s\n", (nbEle*sizeof(double)/1024.0/1024.0)/timer_GPU.GetCounter()); // print speed

    hipMemcpy(out, d_quantArray, sizeof(int)*nbEle, hipMemcpyDeviceToHost);

    free(oriData);
    hipFree(d_oriData);
    hipFree(d_quantArray);
    return 0;
}


int main(int argc, char*argv[])
{
    size_t r1 = 0, r2 = 0, r3 = 0;
    int status = 0;
    char oriFilePath[640], outFilePath[645];
    char qmode[30];
    char type[3];
    if(argc < 3)
    {
        printf("Test case: lorenzoPredQuantCmpr_GPU [type(-f/-d)] [dataFilePath] [quant_mode(QUANT_CODE_ORIGINAL/QUANT_CODE_NORMALIZE] [error_bound] [dims...]\n");
        printf("Example: lorenzoPredQuantCmpr_GPU -f Hurricane.dat QUANT_CODE_ORIGINAL 1E-2 500 500 100\n");
        exit(0);
    }

    sprintf(type, "%s", argv[1]);
    sprintf(oriFilePath, "%s", argv[2]);
    sprintf(outFilePath, "%s.i32", oriFilePath);
    sprintf(qmode, "%s", argv[3]);
    double errorBound = atof(argv[4]);

    if(argc>=6)
        r1 = atoi(argv[5]);
    if(argc>=7)
        r2 = atoi(argv[6]);
    if(argc>=8)
        r3 = atoi(argv[7]);

    int mode = 0;
    if(strcmp(qmode, "QUANT_CODE_ORIGINAL")==0)
        mode = QUANT_CODE_ORIGINAL;
    else if(strcmp(qmode, "QUANT_CODE_NORMALIZE")==0)
        mode = QUANT_CODE_NORMALIZE;
    else
    {
        printf("Error: wrong quantization_code_mode\n");
        exit(0);
    }

    size_t nbEle = computeDataLength(0, 0, r3, r2, r1);
    int* out = (int*)malloc(sizeof(int)*nbEle);
    if(strcmp(type, "-f")==0)
    {
        float* data = readFloatData(oriFilePath, &nbEle, &status);
        status = lorenzoPredictorQuant_Cmpr_NoOutlier_GPU_float(data, mode, errorBound, r3, r2, r1, out);
        free(data);
    }
    else if(strcmp(type, "-d")==0)
    {
        double* data = readDoubleData(oriFilePath, &nbEle, &status);
        status = lorenzoPredictorQuant_Cmpr_NoOutlier_GPU_double(data, mode, errorBound, r3, r2, r1, out);
        free(data);
    }
    else
    {
        printf("Error: wrong data type\n");
        exit(0);
    }

    if(status!=0)
    {
        printf("Error state returned by lorenzoPredictorQuant_GPU function.\n");
	    exit(0);
    }


    writeIntData_inBytes(out, nbEle, outFilePath, &status);
    printf("quantization codes are stored in %s\n", outFilePath);
    free(out);

    return status;
}