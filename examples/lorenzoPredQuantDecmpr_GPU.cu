#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rw.h"
#include "sz_utility.h"

struct PrivateTimingGPU {
    hipEvent_t start;
    hipEvent_t stop;
};

class TimingGPU
{
    private:
        PrivateTimingGPU *privateTimingGPU;

    public:

        TimingGPU();

        ~TimingGPU();

        void StartCounter();
        void StartCounterFlags();

        float GetCounter();

}; // TimingGPU class

// default constructor
TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU;  }

// default destructor
TimingGPU::~TimingGPU() { }

void TimingGPU::StartCounter()
{
    hipEventCreate(&((*privateTimingGPU).start));
    hipEventCreate(&((*privateTimingGPU).stop));
    hipEventRecord((*privateTimingGPU).start,0);
}

void TimingGPU::StartCounterFlags()
{
    int eventflags = hipEventBlockingSync;

    hipEventCreateWithFlags(&((*privateTimingGPU).start),eventflags);
    hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventflags);
    hipEventRecord((*privateTimingGPU).start,0);
}

// Gets the counter in ms
float TimingGPU::GetCounter()
{
    float time;
    hipEventRecord((*privateTimingGPU).stop, 0);
    hipEventSynchronize((*privateTimingGPU).stop);
    hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
    return time;
}


__global__ void kernel_quant_Decmpr_float(float* decData, int* deLorenzoArray, float e2, int bunch)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * bunch;
    int tempIdx;

    for(int i=0; i<bunch; i++)
    {
        tempIdx = index + i;
        decData[tempIdx] = e2 * deLorenzoArray[tempIdx];
    }
}

__global__ void kernel_quant_Decmpr_double(double* decData, int* deLorenzoArray, double e2, int bunch) // yafan reaches here
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * bunch;
    int tempIdx;

    for(int i=0; i<bunch; i++)
    {
        tempIdx = index + i;
        decData[tempIdx] = e2 * deLorenzoArray[tempIdx];
    }
}

int lorenzoPredictorQuant_Decmpr_NoOutlier_GPU_float(int* quantData, int codeFormat, double errorBound, size_t n3, size_t n2, size_t n1, float* result)
{
    TimingGPU timer_GPU;
    size_t nbEle = computeDataLength(0, 0, n3, n2, n1);
    float e2 = errorBound*2;
    int bsize = 256, bunch = 2;
    int gsize = nbEle / (bsize * bunch) + (nbEle % (bsize * bunch) ==0 ? 0 : 1);
    int pad_nbEle = gsize * bsize * bunch;
    int* deLorenzoArray = (int*)malloc(sizeof(int)*nbEle);

    register int curQuantValue = 0;
    register int preQuantValue = 0;
    if(codeFormat == QUANT_CODE_ORIGINAL)
    {
        deLorenzoArray[0] = 0;
        for(size_t i=1; i<nbEle; i++)
        {
            curQuantValue = preQuantValue + quantData[i];
            deLorenzoArray[i] = curQuantValue;
            preQuantValue = curQuantValue;
        }
    }
    else if(codeFormat == QUANT_CODE_NORMALIZE)
    {
        int x = 0;
        deLorenzoArray[0] = 0;
        for(size_t i=1; i<nbEle; i++)
        {
            x = quantData[i];
            curQuantValue = preQuantValue + ((x>>1)^(-(x&1)));
            deLorenzoArray[i] = curQuantValue;
            preQuantValue = curQuantValue;
        }
    }
    else
    {
        printf("Error: wrong quantization_code_mode\n");
        exit(0);
    }

    int* d_deLorenzoArray;
    float* d_result;

    hipMalloc((void**)&d_deLorenzoArray, sizeof(int)*pad_nbEle);
    hipMalloc((void**)&d_result, sizeof(float)*pad_nbEle);
    hipMemcpy(d_deLorenzoArray, deLorenzoArray, sizeof(int)*nbEle, hipMemcpyHostToDevice);

    dim3 blockSize(bsize);
    dim3 gridSize(gsize);

    timer_GPU.StartCounter(); // set timer
    kernel_quant_Decmpr_float<<<gridSize, blockSize>>>(d_result, d_deLorenzoArray, e2, bunch);
    printf("lorenzoPredQuantDecmpr-float speed: %f GB/s\n", (nbEle*sizeof(float)/1024.0/1024.0)/timer_GPU.GetCounter()); // print speed

    hipMemcpy(result, d_result, sizeof(float)*nbEle, hipMemcpyDeviceToHost);

    free(deLorenzoArray);
    hipFree(d_deLorenzoArray);
    hipFree(d_result);

    return 0;
}

int lorenzoPredictorQuant_Decmpr_NoOutlier_GPU_double(int* quantData, int codeFormat, double errorBound, size_t n3, size_t n2, size_t n1, double* result)
{
    TimingGPU timer_GPU;
    size_t nbEle = computeDataLength(0, 0, n3, n2, n1);
    double e2 = errorBound*2;
    int bsize = 256, bunch = 2;
    int gsize = nbEle / (bsize * bunch) + (nbEle % (bsize * bunch) ==0 ? 0 : 1);
    int pad_nbEle = gsize * bsize * bunch;
    int* deLorenzoArray = (int*)malloc(sizeof(int)*nbEle);

    register int curQuantValue = 0;
    register int preQuantValue = 0;
    if(codeFormat == QUANT_CODE_ORIGINAL)
    {
        deLorenzoArray[0] = 0;
        for(size_t i=1; i<nbEle; i++)
        {
            curQuantValue = preQuantValue + quantData[i];
            deLorenzoArray[i] = curQuantValue;
            preQuantValue = curQuantValue;
        }
    }
    else if(codeFormat == QUANT_CODE_NORMALIZE)
    {
        int x = 0;
        deLorenzoArray[0] = 0;
        for(size_t i=1; i<nbEle; i++)
        {
            x = quantData[i];
            curQuantValue = preQuantValue + ((x>>1)^(-(x&1)));
            deLorenzoArray[i] = curQuantValue;
            preQuantValue = curQuantValue;
        }
    }
    else
    {
        printf("Error: wrong quantization_code_mode\n");
        exit(0);
    }

    int* d_deLorenzoArray;
    double* d_result;

    hipMalloc((void**)&d_deLorenzoArray, sizeof(int)*pad_nbEle);
    hipMalloc((void**)&d_result, sizeof(double)*pad_nbEle);
    hipMemcpy(d_deLorenzoArray, deLorenzoArray, sizeof(int)*nbEle, hipMemcpyHostToDevice);

    dim3 blockSize(bsize);
    dim3 gridSize(gsize);

    timer_GPU.StartCounter(); // set timer
    kernel_quant_Decmpr_double<<<gridSize, blockSize>>>(d_result, d_deLorenzoArray, e2, bunch);
    printf("lorenzoPredQuantDecmpr-double speed: %f GB/s\n", (nbEle*sizeof(double)/1024.0/1024.0)/timer_GPU.GetCounter()); // print speed

    hipMemcpy(result, d_result, sizeof(double)*nbEle, hipMemcpyDeviceToHost);

    free(deLorenzoArray);
    hipFree(d_deLorenzoArray);
    hipFree(d_result);

    return 0;
}


int main(int argc, char*argv[])
{
    size_t r1 = 0, r2 = 0, r3 = 0;
    int status = 0;
    char oriFilePath[640], outFilePath[645];
    char qmode[30];
    char type[3];
    if(argc < 3)
    {
        printf("Test case: lorenzoPredQuantDecmpr_GPU [type(-f/-d)] [dataFilePath] [quantization_mode] [error_bound] [dims...]\n");
        printf("Example: lorenzoPredQuantDecmpr_GPU -f Hurricane.dat.bin QUANT_CODE_ORIGINAL 1E-2 500 500 100\n");
	    exit(0);
    }

    sprintf(type, "%s", argv[1]);
    sprintf(oriFilePath, "%s", argv[2]);
    sprintf(outFilePath, "%s.f32", oriFilePath);
    sprintf(qmode, "%s", argv[3]);
    double errorBound = atof(argv[4]);

    if(argc>=6)
        r1 = atoi(argv[5]);
    if(argc>=7)
        r2 = atoi(argv[6]);
    if(argc>=8)
        r3 = atoi(argv[7]);

    int mode = 0;
    if(strcmp(qmode, "QUANT_CODE_ORIGINAL")==0)
        mode = QUANT_CODE_ORIGINAL;
    else if(strcmp(qmode, "QUANT_CODE_NORMALIZE")==0)
        mode = QUANT_CODE_NORMALIZE;
    else
    {
        printf("Error: wrong quantization_code_mode\n");
        exit(0);
    }

    size_t nbEle = 0;
    int *quantData = readInt32Data(oriFilePath, &nbEle, &status);
    nbEle = computeDataLength(0, 0, r3, r2, r1); //update the number of elements
    if(strcmp(type, "-f")==0)
    {
        float* out = (float*)malloc(sizeof(float)*nbEle);
        status = lorenzoPredictorQuant_Decmpr_NoOutlier_GPU_float(quantData, mode, errorBound, r3, r2, r1, out);
    	writeFloatData_inBytes(out, nbEle, outFilePath, &status);
        free(out);
    }
    else
    {
        double* out = (double*)malloc(sizeof(double)*nbEle);
        status = lorenzoPredictorQuant_Decmpr_NoOutlier_GPU_double(quantData, mode, errorBound, r3, r2, r1, out);
    	writeDoubleData_inBytes(out, nbEle, outFilePath, &status);
        free(out);
    }
   
    printf("decompressed data are stored in %s\n", outFilePath);

    free(quantData);
    return status;
}